#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include "matreduce.h"

__global__ void GPU_row_reduce(float* Vgpu, float *aDatagpu, int n, int m)
{
	int i;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < n)
	{
		for (i=m;i--;){
			Vgpu[idx] += fabs(aDatagpu[(idx*m)+i]);
		}
	}
}

__global__ void GPU_col_reduce(float* Vgpu, float* aDatagpu, int n, int m)
{
	int i;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < m)
	{
		for (i=n;i--;){
			Vgpu[idx] += fabs(aDatagpu[idx+(i*n)]);
		}
	}
	
}

__global__ void GPU_sum_vec(float* Vgpu, int n, float *sum)
{
	int i = n/2;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < i)
	{
		sum[idx] = Vgpu[idx*2] + Vgpu[(idx*2)+1];
	}
	else if (idx = i && n%2 == 1)
	{
		sum[idx] = Vgpu[idx];
	}
	i+= n%2;
	while (i!=2)
	{
		i /= 2;
		if (idx < i)
		{
		  sum[idx]+= sum[i-1-idx];
		}		
		i += i%2;	
	}
	if (idx == 0)
	{
		sum[0] += sum[1];
	}
}

extern void run_time_tests(int n, int m, int seed)
{
	int i;
	clock_t begin, end;
	double time_CPUr, time_CPUc, time_GPUr, time_GPUc;
	double time_CPUredrow, time_CPUredcol, time_GPUredrow, time_GPUredcol;
	float *aData, *Arowred, *Acolred;
	float** A;
	
	float *adgpu, *argpu, *acgpu, *rsum_gpu, *csum_gpu;
	
	srand(seed);
	aData = (float*) malloc(sizeof(float)*n*m);
	A = (float**) malloc(sizeof(float*)*n);
	Arowred = (float*) malloc(sizeof(float)*n);
	Acolred = (float*) malloc(sizeof(float)*m);	

	hipMalloc ((void**) &adgpu, sizeof(float)*n*m);
	

	for ( i=0; i<n*m; i++)
	{
		aData[i] =( (float)(drand48()) *2.0 )-1.0;
	}
	for ( i=0; i<n; i++)
	{
		A[i] = &aData[i*m];
	}
	
	memset(Arowred,0.0,sizeof(float)*n);
	memset(Acolred,0.0,sizeof(float)*m);
		
hipMemcpy(acgpu,Acolred,sizeof(float)*m, hipMemcpyHostToDevice);

	int block_size = 8;
	dim3 dimBlock(block_size);
	dim3 dimGrid( (n/dimBlock.x) + (!((n)%dimBlock.x)?0:1) );

	begin = clock();

	CPU_row_reduce(Arowred,A,n,m);

	end = clock();
	
	time_CPUr = (double)(end - begin)/CLOCKS_PER_SEC;

	begin = clock();

	CPU_col_reduce(Acolred,A,n,m);

	end = clock();

	time_CPUc = (double)(end - begin)/CLOCKS_PER_SEC;	

	begin = clock();
	
	hipMalloc ((void**) &argpu, sizeof(float)*n);
	hipMemcpy(adgpu,aData,sizeof(float)*n*m, hipMemcpyHostToDevice);

	GPU_row_reduce<<<dimGrid,dimBlock>>>(argpu,adgpu,n,m);

	hipMemcpy(argpu,Arowred,sizeof(float)*n, hipMemcpyDeviceToHost);

	end = clock();
	
	time_GPUr = (double)(end - begin)/CLOCKS_PER_SEC;

	begin = clock();

	hipMalloc ((void**) &acgpu, sizeof(float)*m);
	GPU_col_reduce<<<dimGrid,dimBlock>>>(acgpu,adgpu,n,m);

	end = clock();
	
	time_GPUc = (double)(end - begin)/CLOCKS_PER_SEC;

	begin = clock();
	
	float total;

	CPU_sum_vec(&total,Arowred,n);

	end = clock();

	time_CPUredrow = (double)(end - begin)/CLOCKS_PER_SEC;


	//hipMalloc ((void**) &sum_gpu, sizeof(float)*n);
	
	printf("Time of CPU on row reduction: %lf\n",time_CPUr);
	printf("Time of CPU on column reduction: %lf\n",time_CPUc);
	printf("Time of GPU on row reduction: %lf\n",time_GPUr);
	printf("Time of GPU on column reduction: %lf\n",time_GPUc);


	free(Arowred);
	free(Acolred);
	free(aData);
	free(A);
	hipFree(argpu);
	hipFree(acgpu);
}


extern void CPU_row_reduce(float* V, float** A, int n, int m )
{
	int i,j;
	for (i = n; i--;)
	{
		for (j = n; j--;)
		{
			V[i] += fabs(A[i][j]);
		}
	}
}

extern void CPU_col_reduce(float* V, float** A, int n, int m)
{
	int i,j;
	for (i=n; i--;)
	{
		for (j=m; j--;)
		{
			V[j] += fabs(A[i][j]);
		}
	}
}

extern void CPU_sum_vec(float* tot, float* V, int n)
{
	int i;
	for (i=0; i<n; i++)
	{
		*tot += V[i];
	}
}





